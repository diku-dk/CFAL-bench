/***********************************************************
 * A Cuda translation of the                               *
 *   faithful implementation of Algorithm 1 from the paper *
 *   proposed by Aaron W. Hsu <arcfide@sacrideo.us>        *
 * Date: May 2024                                          *
 ***********************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#define GPU_RUNS 100

__host__ int gpuAssert(hipError_t code) {
  if(code != hipSuccess) {
    printf("GPU Error: %s\n", hipGetErrorString(code));
    return -1;
  }
  return 0;
}


__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

__global__ void alg1Ker ( int d, int N, int Tc
                        , float* Q, float* K, float* V
                        , float* O, float* ms, float* ls
) {
  // shared memory size: (Bc*(d+1) + 2*Br*d + Br*Bc + 6*Br) * sizeof(float)
  extern __shared__ char sh_mem_char[];

  const int Br = blockDim.y;
  const int Bc = blockDim.x;

  float* Oi   = (float*)sh_mem_char;  // [Br][d]
  float* Qi   = Oi   + Br*d;          // [Br][d]
  float* Pij  = Qi   + Br*d;          // [Br][Bc]
  
  float* Kj   = Pij  + Br*Bc;         // [Bc][d+1]
  float* Vj   = Kj;                   // [Bc][d]

  float* maxs = Kj  + Bc*(d+1);       // [Br]

  float* sums = maxs + Br;            // [Br]
  float* es   = sums + Br;            // [Br]
  float* el   = es   + Br;            // [Br]
  float* li   = el   + Br;            // [Br]
  float* mi   = li   + Br;            // [Br]

  int i  = blockIdx.x;
  int ii = threadIdx.y; // ii < Br
  int jj = threadIdx.x; // jj < Bc

  const int tid = ii*Bc + jj;

  // initialize mi, li
  for (int t = tid; t < Br; t+=Br*Bc) {
    mi[t]   = -INFINITY;
    li[t]   = 0;
  }

  // copy Qi from global to shared memory
  // can be optimized a bit by normalizing the loop
  for (int t = tid; t < Br*d; t+=Br*Bc) {
    int64_t glb_ind = i * Br * d + t;
    Qi[t] = Q[glb_ind];
    Oi[t] = 0;
  }

  __syncthreads();

  for (int j = 0; j < Tc; j++) {

    // initialize maxs, sums
    for (int t = tid; t < Br; t+=Br*Bc) {
      sums[t] = 0;
      maxs[t] = -INFINITY;
    }

    // copy Kj from global to shared memory;
    // can be optimized a bit by normalizing the loop
    // Kj is padded to avoid very expensive bank conflicts in mmm.
    for (int t = tid; t < Bc*d; t+=Br*Bc) {
      int64_t glb_ind = j * Bc * d + t;
      int q = t / d;
      int r = t - q*d; 
      Kj[q*(d+1) + r] = K[glb_ind];
      //Kj[t] = K[glb_ind];
    }
    __syncthreads();

    ////////////////////////////////////
    // first matrix multiplication
    ////////////////////////////////////
    float pij = 0.0;
    {
      for (int kk = 0; kk < d; kk++) {
        pij +=
          Qi[ii * d + kk] *
          Kj[jj*(d+1) + kk]; //Kj[jj * d + kk] ;
      }
    }

    ////////////////////////
    // reductions
    ////////////////////////
    atomicMaxFloat(&maxs[ii], pij);
    __syncthreads();

    {
      pij = exp(pij - maxs[ii]);
      //Pij[ii*Bc + jj] = pij;
      atomicAdd(&sums[ii], pij);
    }
    __syncthreads();

    if(tid < Br) {
      const int ii = tid;
      float mi_old = mi[ii];
      float mx = maxs[ii];
      float mi_new = (mi_old > mx) ? mi_old : mx;
      float eij = exp(mx - mi_new);
      float eli = li[ii] * exp(mi_old - mi_new);
      float li_new = eli + sums[ii] * eij;

      mi[ii] = mi_new;
      li[ii] = li_new;
      es[ii] = eij;
      el[ii] = eli;
    }
    __syncthreads();

    Pij[ii*Bc+jj] = es[ii]*pij;

    // copy Vj from global to shared memory
    for (int t = tid; t < Bc*d; t+=Br*Bc) {
      int64_t glb_ind = j * Bc * d + t;
      Vj[t] = V[glb_ind];
    }
    __syncthreads();

    ////////////////////////////////////
    // second matrix multiplication
    ////////////////////////////////////
    for(int k = 0; k < d / Bc; k++) {
      int kk = k * Bc + jj;
      int ik = ii * d + kk;
      float oi_ik = Oi[ik] * el[ii];

      for(int jjj = 0; jjj < Bc; jjj++) {
        int jk = jjj * d + kk;
        float x = Pij[ii*Bc + jjj];
        float y = Vj[jk];
        oi_ik += x * y;
      }

      Oi[ik] = oi_ik / li[ii];
    }
    __syncthreads();
  }
  
  // copy Oi back to global memory
  for (int t = tid; t < Br*d; t+=Br*Bc) {
    int64_t glb_ind = i * Br * d + t;
    O[glb_ind] = Oi[t];
  }

  // copy ms, ls back to global memory
  for (int t = tid; t < Br; t+=Br*Bc) {
    int64_t glb_ind = i * Br + t;
    ms[glb_ind] = mi[t];
    ls[glb_ind] = li[t];
  }
}

////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////

__host__ float
L2(float *x, size_t c)
{
        float sum;

        sum = 0;

        for (size_t i = 0; i < c; i++)
                sum += x[i] * x[i];

        return sqrt(sum);
}

__host__ int
flash_attention(float* m_d, float* l_d, float *O_d, float *Q_d, float *K_d, float *V_d, int N, int d, int M)
{
    int Br, Bc, Tr, Tc;

    Bc = M / (4 * d);
    Br = d < Bc ? d : Bc;
    Tr = N / Br;
    Tc = N / Bc;

    // setup execution parameters
    dim3 block(Bc, Br, 1);
    dim3 grid (Tr,  1, 1);
    const size_t shmem_size = (Bc*(d+1) + 2*Br*d + Br*Bc + 6*Br) * sizeof(float);
    //printf("\nShared memory size: %d, Bc=%d, Br=%d, Tc: %d, Tr: %d, d: %d, N: %d\n", shmem_size, Bc, Br, Tc, Tr, d, N);

    alg1Ker<<<grid, block, shmem_size>>>( d, N, Tc, Q_d, K_d, V_d, O_d, m_d, l_d );
    //cudaDeviceSynchronize();

#if 0
    {
        float* m_h = (float*) malloc(N*sizeof(float));
        cudaMemcpy(m_h, m_d, N*sizeof(float), cudaMemcpyDeviceToHost);
        {
            printf("(N,d,Br,Bc,Tr,Tc)=(%d,%d,%d,%d,%d,%d), ms:\n    ", N,d,Br,Bc,Tr,Tc);
            for(int q=0; q<Br; q++) {
                printf(", %f", m_h[q]);
            }
            printf("\n");
        }

        free(m_h);
    }
#endif

    return 0;
}

int main(int argc, char **argv)
{
        
    struct timeval ts, te;
    size_t cnt;
    float *Q, *K, *V, *O, dur;
    int N, d, M;

    if (argc != 3 && argc != 4) {
        fprintf(stderr, "Usage:\n");
        fprintf(stderr, "  %s d N M  Compute with matrices filled with ones\n", argv[0]);
        fprintf(stderr, "  %s M -io  Read matrices from stdin and write O to stdout\n", argv[0]);
        return EXIT_FAILURE;
    }

    bool io_arrays = false;
    if (argc == 3) {
        if (strcmp(argv[2], "-io") != 0) {
                fprintf(stderr, "Invalid argument '%s'\n", argv[1]);
                return EXIT_FAILURE;
        }
        io_arrays = true;
    }

    if (io_arrays) {
        M = atoi(argv[1]);
        scanf("%d %d", &d, &N);
    } else {
        d = atoi(argv[1]);
        N = atoi(argv[2]);
        M = atoi(argv[3]);
    }

    if (N % d != 0) {
        fprintf(stderr, "d must divide N\n");
        return EXIT_FAILURE;
    }

    cnt = N * d;

    fprintf(stderr, "Initializing data...");

    if ((Q = (float*)calloc(cnt, sizeof(float))) == NULL)
        goto mem_failure;

    if ((K = (float*)calloc(cnt, sizeof(float))) == NULL)
        goto mem_failure;

    if ((V = (float*)calloc(cnt, sizeof(float))) == NULL)
        goto mem_failure;

    if ((O = (float*)calloc(cnt, sizeof(float))) == NULL)
        goto mem_failure;

    if (io_arrays) {
        for (int i = 0; i < d * N; i++) scanf("%f", &Q[i]);
        for (int i = 0; i < d * N; i++) scanf("%f", &K[i]);
        for (int i = 0; i < d * N; i++) scanf("%f", &V[i]);
    } else {
        for (size_t i = 0; i < cnt; i++) {
            Q[i] = K[i] = V[i] = 1;
        }
    }

    fprintf(stderr, "done.\n");

    {
        float *Q_d, *K_d, *V_d, *O_d, *m_d, *l_d;

        hipSetDevice(1);

        // allocate memory on device
        hipMalloc((void**) &Q_d, cnt*sizeof(float));
        hipMalloc((void**) &O_d, cnt*sizeof(float));
        hipMalloc((void**) &V_d, cnt*sizeof(float));
        hipMalloc((void**) &K_d, cnt*sizeof(float));
        hipMalloc((void**) &m_d, N*sizeof(float));
        hipMalloc((void**) &l_d, N*sizeof(float));

        // copy host memory to device
        hipMemcpy(Q_d, Q, cnt*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(V_d, V, cnt*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(K_d, K, cnt*sizeof(float), hipMemcpyHostToDevice);

        fprintf(stderr, "Warming up...");

        flash_attention(m_d, l_d, O_d, Q_d, K_d, V_d, N, d, M);
        hipDeviceSynchronize();
        gpuAssert( hipPeekAtLastError() );

        fprintf(stderr, "done.\n");
        fprintf(stderr, "Running flash_attention...");

        hipFuncSetAttribute(reinterpret_cast<const void*>(alg1Ker), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

        gettimeofday(&ts, NULL);

        for(int i=0; i<GPU_RUNS; i++) {
            flash_attention(m_d, l_d, O_d, Q_d, K_d, V_d, N, d, M);
        }
        hipDeviceSynchronize();
        gpuAssert( hipPeekAtLastError() );

        gettimeofday(&te, NULL);

        fprintf(stderr, "done.\n");

        dur = (double)(te.tv_usec - ts.tv_usec) / 1e6 +
                          (double)(te.tv_sec - ts.tv_sec);
        dur = dur / GPU_RUNS;

        hipMemcpy(O, O_d, cnt*sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        if (io_arrays) {
            for (int i = 0; i < N; i++) {
                for (int j = 0; j < d; j++) {
                    if (j > 0) putchar(' ');
                    printf("%f", O[d * i + j]);
                }
                putchar('\n');
            }
        } else {
            fprintf(stderr, "L2 norm is %lf (should be %lf)\n", L2(O, cnt), sqrt(cnt));
        }

        /* QK^t is 2N^2d flops, so is PV. softmax(S) (row-wise)
         * exp(S[i]) / sum_j exp(P[i, j] - max(P[i]))
         * is N * (N + 4N) = 5 N^2 flops, but exp is more expensive. */
        fprintf(stderr,
                "Compute rate: %lf Gflops/s, runtime: %lf\n",
                (4.0 * d + 5.0) * N * N / dur / 1e9,  dur*1e6);

        hipFree(Q_d);
        hipFree(K_d);
        hipFree(V_d);
        hipFree(O_d);
        hipFree(m_d);
        hipFree(l_d);
    }

    free(Q);
    free(K);
    free(V);
    free(O);

    return EXIT_SUCCESS;

mem_failure:
        printf("Failed to allocate memory.\n");
        return EXIT_FAILURE;
}
